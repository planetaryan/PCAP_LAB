#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void multiply(int*a,int*b,int*c){
    int index=threadIdx.x;
    c[index]=a[index]*b[index];
}

int main(){

    int n=5;
    int size=n*sizeof(int);

    int h_a[]={1,2,3,4,5};
    int h_b[]={2,3,4,5,6};
    int h_c[n];

    int *d_a,*d_b,*d_c;

    hipMalloc(&d_a,size);
    hipMalloc(&d_b,size);
    hipMalloc(&d_c,size);

    hipMemcpy(d_a,h_a,size,hipMemcpyHostToDevice);
    hipMemcpy(d_b,h_b,size,hipMemcpyHostToDevice);
    
    multiply<<<1,n>>>(d_a,d_b,d_c);

    hipMemcpy(h_c,d_c,size,hipMemcpyDeviceToHost);

    printf("Result: ");
    for(int i=0;i<n;i++){
        printf("%d ",h_c[i]);
    }
    printf("\n");
    return 0;

}